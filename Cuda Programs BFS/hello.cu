#include <stdio.h>
#include <hip/hip_runtime.h>

void CPUFunction()
{
  printf("Runs on the CPU.\n");
}

__global__
void GPUFunction()
{
  printf("Runs on the GPU.\n");
}

int main()
{
  CPUFunction();

  GPUFunction<<<1, 1>>>();

  hipDeviceSynchronize();

  return EXIT_SUCCESS;
}