#include "hip/hip_runtime.h"

#include<math.h>
#include <stdio.h>
#include<time.h>
#include <iostream>
#include <stdlib.h>

using namespace std;

#define BLOCK_SIZE 16

void matMulCPU(float* A, float* B, float* C, int numARows, int numACols, int numBCols) {
    int i, j, k;
    int offsetA, offsetB;
    float cumSum;

    for (i = 0; i < numARows; i++) {
        for (j = 0; j < numBCols; j++) {
            cumSum = 0;
            for (k = 0; k < numACols; k++) {
                // linearize index
                offsetA = i*numACols + k;
                offsetB = k*numBCols + j;

                // accumulate element-wise product
                cumSum += A[offsetA] * B[offsetB];
            }
            C[i*numBCols + j] = cumSum;
        }
    }
}

__global__ void matMulGPU(float* A, float* B, float* C, int numARows, int numACols, int numBCols) {
    // compute global thread coordinates
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // linearize coordinates for data access
    int offset = row * numBCols + col;

    if ((row < numARows) && (col < numBCols)) {
        float cumSum = 0;
        for (int k = 0; k < numACols; k++) {
            cumSum += A[row*numACols + k] * B[k*numBCols + col];
        }
        C[offset] = cumSum;
    }
}

int main(void) {

    int numARows = 960;
    int numACols = 640;
    int numBCols = 800;
    size_t sizeA = numARows * numACols * sizeof(float);
    size_t sizeB = numACols * numBCols * sizeof(float);
    size_t sizeC = numARows * numBCols * sizeof(float);

    // allocate host memory
    float* h_A = (float*)malloc(sizeA);
    float* h_B = (float*)malloc(sizeB);
    float* h_C = (float*)malloc(sizeC);
    float* h_C_CPU = (float*)malloc(sizeC);

    // initialize host matrices
    int i, j, offset;
    for (i = 0; i <  numARows; i++) {
        for (j = 0; j < numACols; j++) {
            offset = i*numACols + j;
            h_A[offset] = sin(i);
        }
    }
    for (i = 0; i <  numACols; i++) {
        for (j = 0; j < numBCols; j++) {
            offset = i*numBCols + j;
            h_B[offset] = cos(j);
        }
    }

    // allocate device matrices
    float* d_A;
    float* d_B;
    float* d_C;
    hipMalloc((void**)&d_A, sizeA);
    hipMalloc((void**)&d_B, sizeB);
    hipMalloc((void**)&d_C, sizeC);

    // transfer to GPU
    hipMemcpy(d_A, h_A, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, sizeB, hipMemcpyHostToDevice);

    // kernel launch
    dim3 threadPerBlock(BLOCK_SIZE, BLOCK_SIZE, 1);
    dim3 blockPerGrid(ceil(numBCols/(float)BLOCK_SIZE), ceil(numACols/(float)BLOCK_SIZE), 1);
    matMulGPU<<<blockPerGrid, threadPerBlock>>>(d_A, d_B, d_C, numARows, numACols, numBCols);

    // transfer to CPU
    hipMemcpy(h_C, d_C, sizeC, hipMemcpyDeviceToHost);

    clock_t begin = clock();
    matMulCPU(h_A, h_B, h_C_CPU, numARows, numACols, numBCols);
    clock_t end = clock();
    double time_spent = (double)(end - begin) / CLOCKS_PER_SEC * 1000;
    printf("Implemented CPU serial code ran in: %f msecs.\n", time_spent);

    // verify correctness of results
    for (i=0; i<numACols; i++)
        for (j=0; j<numBCols; j++)
            if (fabs(h_C_CPU[i*numBCols+j] - h_C[i*numBCols+j]) > 1e-2) {
                fprintf(stderr, "Result verification failed at element (%d,%d)!\n", i, j);
                exit(EXIT_FAILURE);
            }
    printf("Test PASSED\n");

    free(h_A); free(h_B); free(h_C); free(h_C_CPU);
    hipFree(d_A); hipFree(d_B); hipFree(d_C);

    return 0;
}